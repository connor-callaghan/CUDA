// Multiply matrices using the BLAS library

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <assert.h>
#include <time.h>

// Verify results on the CPU
void verify_result(float *a, float *b, float *c, int n)
{
    float temp;
    float epsilon = 0.001;
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            temp = 0;
            for (int k = 0; k < n; k++)
            {
                // Translate to column major order based on cuBLAS memory layout
                temp += a[k * n + i] * b[j * n + k];
            }
            assert(fabs(c[j * n + i] - temp) < epsilon);
        }
    }
}

int main()
{
    // Define Size of Problem
    int n = 1 << 10;
    size_t bytes = n * n * sizeof(float);

    // Declare matrix pointers on host and device
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    // Allocate memory for above pointers
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Pseudo random number generator with a reusable seed, Using curand library
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

    // Initialize matrix with random numbers on the device
    hiprandGenerateUniform(prng, d_a, n*n);
    hiprandGenerateUniform(prng, d_b, n*n);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Scaling Factors
    float alpha = 1.0f;
    float beta = 0.0f;

    // Calculation: C = (alpha*A) * B + (beta*C)
    // (m x n) * (n * k) = (m X k)
    // Signature for cublasSgemm: handle, operation (use CUBLAS_OP_T for transposing), operation, m, n, k, address of alpha, A, lda, B, ldb, address of beta, C, ldc
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_a, n, d_b, n, &beta, d_c, n);

    // Copy back to host
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // verify result
    verify_result(h_a, h_b, h_c, n);

    printf("SUCCESS\n");

    return 0;
}